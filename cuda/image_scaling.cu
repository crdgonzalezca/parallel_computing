#include "hip/hip_runtime.h"
// #include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
# include <opencv2/opencv.hpp>
#include <bits/stdc++.h>
#include <sys/time.h>

using namespace cv;
using namespace std;

#define RESULT_WIDTH 720
#define RESULT_HEIGHT 480
#define ITERATIONS 10
#define MS 1000000.0

typedef unsigned long long timestamp_t;

timestamp_t get_timestamp (){
    struct timeval now;
    gettimeofday (&now, NULL);
    return now.tv_usec + (timestamp_t)now.tv_sec * 1000000;
}

/**
 * CUDA Kernel Device code
 *
 * Computes the new scaled output_image with NNS algorithm.
 */
__global__ void nearest_neighbour_scaling(
    unsigned char *input_image, 
    unsigned char *output_image,
    int width_input, 
    int height_input,
    int channels_input,
    int width_output, 
    int height_output,
    int channels_output) {
    const float x_ratio = (width_input + 0.0) / width_output;
    const float y_ratio = (height_input + 0.0) / height_output;

    //2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    int px = 0, py = 0; 
    const int input_width_step = width_input * channels_input;
    const int output_width_step = width_output * channels_output;

    if ((xIndex < width_output) && (yIndex < height_output)){
        py = ceil(yIndex * y_ratio);
        px = ceil(xIndex * x_ratio);
        for (int channel = 0; channel < channels_output; channel++){
            *(output_image + (yIndex * output_width_step + xIndex * channels_output + channel)) =  *(input_image + (py * input_width_step + px * channels_input + channel));
        }
    }
}

/**
Implementation of Bilinear interpolation algorithm to down 
sample the source image.
*/
__global__ void bilinear_scaling(
    unsigned char *input_image, 
    unsigned char *output_image,
    int width_input, 
    int height_input,
    int channels_input,
    int width_output, 
    int height_output,
    int channels_output) {

    const float x_ratio = (width_input + 0.0) / width_output;
    const float y_ratio = (height_input + 0.0) / height_output;

    //2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    const int input_width_step = width_input * channels_input;
    const int output_width_step = width_output * channels_output;

    if ((xIndex < width_output) && (yIndex < height_output)){
        int py = (int)(yIndex * y_ratio);
        int px = (int)(xIndex * x_ratio);
    
        float x_diff = (x_ratio * xIndex) - px;
        float y_diff = (y_ratio * yIndex) - py;
    
        uchar *ptr_img = input_image + (py * input_width_step);
        uchar *ptr_img_2 = input_image + (py * (input_width_step + 1));

        for (int channel = 0; channel < channels_target; channel++){
            int column = channels_input * px + channel;

            int pixel_value = *(ptr_img + column) * (1 - x_diff) * (1 - y_diff) +
                    *(ptr_img + column + channels_input) * x_diff * (1 - y_diff) +
                    *(ptr_img_2 + column) * (1 - x_diff) * y_diff + 
                    *(ptr_img_2 + column + channels_input) * x_diff * y_diff;
            *(output_image + (yIndex * output_width_step + xIndex * channels_output + channel)) = pixel_value;
        }
    }
}

/**
 * Host main routine
 */
int main(int argc, char* argv[]) {
    // Read parameters 1- source path, 2- Destination path, 3- algorithm
    if (argc != 5) {
        cout << "Arguments are not complete. Usage: image_path image_result_path n_threads algorithm" << endl;
        return 1;
    }
    const string source_image_path = argv[1];
    const string result_image_path = argv[2];
    const int threads = atoi(argv[3]);
    const string algorithm = argv[4];

    // time measurement variables
    hipEvent_t start, end;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Create result image of 720x480 pixels with 3 channels
    Mat output_image(RESULT_HEIGHT, RESULT_WIDTH, CV_8UC3, Scalar(255, 255, 255)); 
    // Read the image from the given source path
    Mat input_image = imread(source_image_path);
    if(input_image.empty()) {
        printf("Error reading image.");
        return 1;
    }
    
    // Matrices sizes width * height * 3
    const int input_bytes = input_image.cols * input_image.rows * input_image.channels() * sizeof(unsigned char);
    const int output_bytes = output_image.cols * output_image.rows * output_image.channels() * sizeof(unsigned char);

    unsigned char *d_input, *d_output;
    // Allocate the device input image
    err = hipMalloc<unsigned char>(&d_input, input_bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device input image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc<unsigned char>(&d_output, output_bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device output image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input image in host memory to the device input image in device memory
    err = hipMemcpy(d_input, input_image.ptr(), input_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy input image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipEventCreate(&start);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipEventCreate(&end);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int width_input = input_image.cols;
    int height_input = input_image.rows;
    int channels_input = input_image.channels();
    int width_output = output_image.cols;
    int height_output = output_image.rows;
    int channels_output = output_image.channels();

    const dim3 threadsPerBlock(threads, threads);
    //Calculate numBlocks size to cover the whole image        
    const dim3 numBlocks(width_output / threadsPerBlock.x, height_output / threadsPerBlock.y);

    for(int i = 0; i < ITERATIONS; i++){
        
        if(algorithm == "Nearest") {
            nearest_neighbour_scaling<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width_input, height_input, channels_input, width_output, height_output, channels_output);
        } else if(algorithm == "Bilinear") {
            bilinear_scaling<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width_input, height_input, channels_input, width_output, height_output, channels_output);
        }
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // Record the stop event
    err = hipEventRecord(end, NULL);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    err = hipEventSynchronize(end);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, end);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float secPerMatrixMul = 1e-3 * msecTotal / ITERATIONS;
    double flopsPerMatrixMul = 2.0 * (double)width_output * (double)height_output * channels_output;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (secPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.8f s, Size= %.0f Ops, WorkgroupSize= %u threads/block, Blocks= %u\n",
        gigaFlops,
        secPerMatrixMul,
        flopsPerMatrixMul,
        threadsPerBlock.x * threadsPerBlock.y,
        numBlocks.x * numBlocks.y
    );

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    err = hipMemcpy(output_image.ptr(), d_output, output_bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Write the image to a file
    imwrite(result_image_path, output_image);

    // Free device global memory
    err = hipFree(d_input);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Done\n");
    return 0;
}

