#include "hip/hip_runtime.h"
// #include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
# include <opencv2/opencv.hpp>
#include <bits/stdc++.h>
#include <sys/time.h>

using namespace cv;
using namespace std;

#define RESULT_WIDTH 720
#define RESULT_HEIGHT 480
#define ITERATIONS 10
#define MS 1000000.0

typedef unsigned long long timestamp_t;

timestamp_t get_timestamp (){
    struct timeval now;
    gettimeofday (&now, NULL);
    return now.tv_usec + (timestamp_t)now.tv_sec * 1000000;
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(unsigned char *input_image, unsigned char *output_image, 	int width, int height, 
    int inputWidthStep, int outputWidthStep){
    //2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if ((xIndex < width) && (yIndex < height)){
        const int color_tid = yIndex * outputWidthStep + (3 * xIndex);

		//Location of gray pixel in output
		// const int gray_tid = yIndex * outputWidthStep + xIndex;

		const unsigned char blue = input_image[color_tid];
		const unsigned char green = input_image[color_tid + 1];
		const unsigned char red = input_image[color_tid + 2];

		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

        output_image[color_tid] = static_cast<unsigned char>(gray);
        output_image[color_tid + 1] = static_cast<unsigned char>(gray);
        output_image[color_tid + 2] = static_cast<unsigned char>(gray);
    }
}

__global__ void nearest_neighbour_scaling(
    unsigned char *input_image, 
    unsigned char *output_image,
    int width_input, 
    int height_input,
    int channels_input,
    int width_output, 
    int height_output,
    int channels_output) {
    const float x_ratio = (width_input + 0.0) / width_output;
    const float y_ratio = (height_input + 0.0) / height_output;

    //2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    //if ((xIndex < 100) && (yIndex < 100))
    //    printf("%d - %d\n", xIndex, yIndex);

    int px = 0, py = 0; 
    int input_width_step = width_input * channels_input;
    int output_width_step = width_output * channels_output;
    if ((xIndex < width_output) && (yIndex < height_output)){
        py = ceil(yIndex * y_ratio);
        px = ceil(xIndex * x_ratio);
            //ptr_source = img.ptr<uchar>(py);
            
            // Calculate the value of the i,j pixel for each channel
        for (int channel = 0; channel < channels_output; channel++){
            *(output_image + (yIndex * output_width_step + xIndex * channels_output + channel)) =  *(input_image + (py * input_width_step + px * channels_output +  + channel));
        }
    }
}


/**
 * Host main routine
 */
int main(int argc, char* argv[]) {
    // Read parameters 1- source path, 2- Destination path, 3- Number of threads, 4- algorithm
    if (argc != 4) {
        cout << "Arguments are not complete. Usage: image_path image_result_path n_threads algorithm" << endl;
        return 1;
    }
    string source_image_path = argv[1];
    string result_image_path = argv[2];
    // THREADS = atoi(argv[3]);
    string algorithm = argv[3];

    // time measurement variables
    timestamp_t start, end;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Create result image of 720x480 pixels with 3 channels
    Mat output_image(RESULT_HEIGHT, RESULT_WIDTH, CV_8UC3, Scalar(255, 255, 255)); 
    // Read the image from the given source path
    Mat input_image = imread(source_image_path);
    if(input_image.empty()) {
        printf("Error reading image.");
        return 1;
    }

    // Matrices sizes width * height * 3
    const int input_bytes = input_image.cols * input_image.rows * input_image.channels() * sizeof(unsigned char);
    const int output_bytes = output_image.cols * output_image.rows * output_image.channels() * sizeof(unsigned char);

    unsigned char *d_input, *d_output;
    // Allocate the device input image
//    float *d_A = NULL;
    err = hipMalloc<unsigned char>(&d_input, input_bytes);
//    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device input image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output image
//    float *d_B = NULL;
    err = hipMalloc<unsigned char>(&d_output, output_bytes);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device output image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_input, input_image.ptr(), input_bytes, hipMemcpyHostToDevice);
//    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy input image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_output, output_image.ptr(), output_bytes, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy output image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }    
    
    //int threadsPerBlock = 256;
    // int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    int width_input = input_image.cols;
    int height_input = input_image.rows;
    int channels_input = input_image.channels();
    int width_output = output_image.cols;
    int height_output = output_image.rows;
    int channels_output = output_image.channels();

    // Launch the Vector Add CUDA Kernel
    const dim3 threadsPerBlock(16, 16);

	//Calculate numBlocks size to cover the whole image
    const dim3 numBlocks((width_output + threadsPerBlock.x - 1) / threadsPerBlock.x, (height_output + threadsPerBlock.y - 1) / threadsPerBlock.y);
    nearest_neighbour_scaling<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width_input, height_input, channels_input, width_output, height_output, channels_output);
//    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(output_image.ptr(), d_output, output_bytes, hipMemcpyDeviceToHost);
    // err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_input);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    imwrite(result_image_path, output_image); //write the image to a file

    printf("Done\n");
    return 0;
}

