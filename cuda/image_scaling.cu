#include "hip/hip_runtime.h"
// #include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <opencv2/opencv.hpp>
#include <bits/stdc++.h>
#include <sys/time.h>

using namespace cv;
using namespace std;

#define RESULT_WIDTH 720
#define RESULT_HEIGHT 480
#define ITERATIONS 20

// Function taken from https://github.com/sshniro/opencv-samples/blob/master/cuda-bgr-grey.cpp
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number) {
	if (err != hipSuccess) {
		fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg, file_name, line_number, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

/**
 * CUDA Kernel Device code
 *
 * Computes the new scaled output_image with NNS algorithm.
 */
__global__ void nearest_neighbour_scaling(
    unsigned char *input_image, 
    unsigned char *output_image,
    int width_input, 
    int height_input,
    int channels_input,
    int width_output, 
    int height_output,
    int channels_output) {
    const float x_ratio = (width_input + 0.0) / width_output;
    const float y_ratio = (height_input + 0.0) / height_output;

    //2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    int px = 0, py = 0; 
    const int input_width_step = width_input * channels_input;
    const int output_width_step = width_output * channels_output;

    if ((xIndex < width_output) && (yIndex < height_output)){
        py = ceil(yIndex * y_ratio);
        px = ceil(xIndex * x_ratio);
        for (int channel = 0; channel < channels_output; channel++){
            *(output_image + (yIndex * output_width_step + xIndex * channels_output + channel)) =  *(input_image + (py * input_width_step + px * channels_input + channel));
        }
    }
}

/**
* CUDA Kernel Device code
*
* Implementation of Bilinear interpolation algorithm to down sample the source image.
*/
__global__ void bilinear_scaling(
    unsigned char *input_image, 
    unsigned char *output_image,
    int width_input, 
    int height_input,
    int channels_input,
    int width_output, 
    int height_output,
    int channels_output) {

    const float x_ratio = (width_input + 0.0) / width_output;
    const float y_ratio = (height_input + 0.0) / height_output;

    //2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    const int input_width_step = width_input * channels_input;
    const int output_width_step = width_output * channels_output;

    if ((xIndex < width_output) && (yIndex < height_output)){
        int py = (int)(yIndex * y_ratio);
        int px = (int)(xIndex * x_ratio);
    
        float x_diff = (x_ratio * xIndex) - px;
        float y_diff = (y_ratio * yIndex) - py;
    
        uchar *ptr_img = input_image + (py * input_width_step);
        uchar *ptr_img_2 = input_image + ((py + 1) * input_width_step);

        for (int channel = 0; channel < channels_input; channel++){
            int column = channels_input * px + channel;

            int pixel_value = *(ptr_img + column) * (1 - x_diff) * (1 - y_diff) +
                    *(ptr_img + column + channels_input) * x_diff * (1 - y_diff) +
                    *(ptr_img_2 + column) * (1 - x_diff) * y_diff + 
                    *(ptr_img_2 + column + channels_input) * x_diff * y_diff;
            *(output_image + (yIndex * output_width_step + xIndex * channels_output + channel)) = pixel_value;
        }
    }
}

/**
 * Host main routine
 */
int main(int argc, char* argv[]) {
    // Read parameters 1- source path, 2- Destination path, 3-threads, 4- algorithm
    if (argc != 5) {
        printf("Arguments are not complete. Usage: image_path image_result_path n_threads algorithm.\n");
        exit(EXIT_FAILURE);
    }
    const string source_image_path = argv[1];
    const string result_image_path = argv[2];
    const int threads = atoi(argv[3]);
    const string algorithm = argv[4];

    // time measurement variables
    hipEvent_t start, end;

    // Create result image of 720x480 pixels with 3 channels
    Mat output_image(RESULT_HEIGHT, RESULT_WIDTH, CV_8UC3, Scalar(255, 255, 255)); 
    // Read the image from the given source path
    Mat input_image = imread(source_image_path);
    if(input_image.empty()) {
        printf("Error reading image.");
        exit(EXIT_FAILURE);
    }
    
    // Matrices sizes width * height * 3
    const int input_bytes = input_image.cols * input_image.rows * input_image.channels() * sizeof(unsigned char);
    const int output_bytes = output_image.cols * output_image.rows * output_image.channels() * sizeof(unsigned char);

    unsigned char *d_input, *d_output;
    // Allocate the device input image
    SAFE_CALL(hipMalloc<unsigned char>(&d_input, input_bytes), "Failed to allocate device input image.");
    // Allocate the device output image
    SAFE_CALL(hipMalloc<unsigned char>(&d_output, output_bytes), "Failed to allocate device output image.");

    // Copy the host input image in host memory to the device input image in device memory
    SAFE_CALL(hipMemcpy(d_input, input_image.ptr(), input_bytes, hipMemcpyHostToDevice), "Failed to copy input image from host to device");

    // Create event to measure start time
    SAFE_CALL(hipEventCreate(&start), "Failed to create start event.");

    // Create event to measure end time
    SAFE_CALL(hipEventCreate(&end), "Failed to create end event");

    // Record the start event
    SAFE_CALL(hipEventRecord(start, NULL), "Failed to start rescor of start event");
    
    int width_input = input_image.cols;
    int height_input = input_image.rows;
    int channels_input = input_image.channels();
    int width_output = output_image.cols;
    int height_output = output_image.rows;
    int channels_output = output_image.channels();

    const dim3 threadsPerBlock(threads, threads);
    //Calculate numBlocks size to cover the whole image        
    const dim3 numBlocks(width_output / threadsPerBlock.x, height_output / threadsPerBlock.y);

    // Run kernel several times to measure an average time.
    for(int i = 0; i < ITERATIONS; i++){
        if(algorithm == "Nearest") {
            nearest_neighbour_scaling<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width_input, height_input, channels_input, width_output, height_output, channels_output);
        } else if(algorithm == "Bilinear") {
            bilinear_scaling<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width_input, height_input, channels_input, width_output, height_output, channels_output);
        }
        SAFE_CALL(hipGetLastError(), "Failed to launch kernel");
    }

    // Record the stop event
    SAFE_CALL(hipEventRecord(end, NULL), "Failed to record end event.");

    // Wait for the stop event to complete
    SAFE_CALL(hipEventSynchronize(end), "Failed to synchronize on the end event");

    float msecTotal = 0.0f;
    SAFE_CALL(hipEventElapsedTime(&msecTotal, start, end), "Failed to get time elapsed between events");

    // Compute and print the performance
    float secPerMatrixMul = msecTotal / (ITERATIONS * 1000.0f);
    double flopsPerMatrixMul = 2.0 * (double)width_output * (double)height_output * channels_output;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (secPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.8f s, Size= %.0f Ops, WorkgroupSize= %u threads/block, Blocks= %u\n",
        gigaFlops,
        secPerMatrixMul,
        flopsPerMatrixMul,
        threadsPerBlock.x * threadsPerBlock.y,
        numBlocks.x * numBlocks.y
    );

    // Copy the device output image in device memory to the host output image in host memory.
    SAFE_CALL(hipMemcpy(output_image.ptr(), d_output, output_bytes, hipMemcpyDeviceToHost), "Failed to copy output image from device to host");

    // Write the image to a file
    imwrite(result_image_path, output_image);

    // Free device global memory
    SAFE_CALL(hipFree(d_input), "Failed to free device input image");
    SAFE_CALL(hipFree(d_output), "Failed to free device output image");

    printf("Done\n");
    return 0;
}

